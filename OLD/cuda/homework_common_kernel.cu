#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
void random_ints(int* a, int N)
{
   int i;
   for (i = 0; i < N; ++i)
    a[i] = rand();
}
__global__ void anyStep(int *in, int *out, int size, int step){
    int temp;
    int idx = threadIdx.x + (blockDim.x * blockIdx.x) ;
    if(idx % 2 == step && idx < size -1){
        if(in[idx]>in[idx+1]){
            temp=in[idx];
            in[idx]=in[idx+1];
            in[idx+1]=temp;
        }
    }f

    __syncthreads();
    if (idx < size)
        out[idx]=in[idx];
}
int main(void){
    int i;
    int *a,*a_sorted;
    int *d_a,*d_sorted;
    int blocks=6;
    int threads = 1024;
    int n=blocks*threads;
    int size =sizeof(int) * n;
    hipMalloc((void**) &d_a,size);
    hipMalloc((void**) &d_sorted,size);
    a   = (int*) malloc(size); random_ints(a,n);
    a_sorted = (int*) malloc(size);

    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    for(i=1;i<=n/2;i++){
        anyStep<<<blocks,threads>>>(d_a,d_sorted,n,1);
        
        anyStep<<<blocks,threads>>>(d_sorted,d_a,n,0);
        
    }
    hipMemcpy(a_sorted,d_a,size,hipMemcpyDeviceToHost);
    for(i=0;i<n;i++){
        printf("%d\t",a_sorted[i]);
    }
    return 0;
}