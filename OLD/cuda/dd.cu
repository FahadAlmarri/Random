#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <ctype.h>

#include <stdlib.h>
#include <string.h>
__global__ void
rank_sort(int N, int input[], int output[]) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    

    if (index < N)
    {
        
         for (int i = 0; i <N ; i++) {

        int k=0;
        for (int j=0;j<N;j++)
        {
            if (input[index]>input[j])
            {
                k++;
            }
            
        }
        
        {
        if(output[k]==input[index]){
             output[k+1]=input[index];
             }
        else{
            
            output[k]=input[index];
        }

            
        }
        
         }  
    }
    

}


int main(void) {
 int * a, * a_sorted;
 int * d_a, * d_sorted;
 int n = 8;
 int size = sizeof(int) * n;
 int blocks = 10;
 int threads = 10;
 hipMalloc((void ** ) & d_a, size);
 hipMalloc((void ** ) & d_sorted, size);
 a = (int * ) malloc(size);
 a_sorted = (int * ) malloc(size);
printf("Enter the unsorted numbers:\n");
    for(int i=0; i<n; i++){
        scanf("%d", &a[i]);
    }



 printf(" unsorted numbers: \t");1
 for (int i = 0; i < n; i++) {
 printf("%d\t", a[i]);
 }
 hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
 rank_sort << < blocks, threads >>> (n, d_a,d_sorted);

 
 hipMemcpy(a_sorted, d_sorted, size, hipMemcpyDeviceToHost);
 printf(" Sorted numbers :\t");
 for (int i = 0; i < n; i++) {
 printf("%d\t", a_sorted[i]);
 }
 free(a);
 free(a_sorted);
 hipFree(d_a);
 hipFree(d_sorted);
 return 0;
}
