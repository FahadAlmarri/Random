#include <stdio.h>
#include <hip/hip_runtime.h>


void random_ints(int* a, int N)
{
   int i;
   for (i = 0; i < N; ++i)
    a[i] = rand();
}
__global__ void loop_inside(int *in, int *out, int size){
    __shared__ bool swapedodd;
    __shared__ bool swapedeven;
    bool odd = true;
    int temp;
    int idx = threadIdx.x ;
    while(true){
        __syncthreads();
        if(odd == true){
            swapedodd = false;
            __syncthreads();
            if(idx % 2 == 1 && idx < size -1){
                if(in[idx]>in[idx+1]){
                    temp=in[idx];
                    in[idx]=in[idx+1];
                    in[idx+1]=temp;
                    swapedodd = true;
                }
            }
        }
        else{
            swapedeven = false;
            __syncthreads();
             if(idx % 2 == 0 && idx < size -1){
                if(in[idx]>in[idx+1]){
                    temp=in[idx];
                    in[idx]=in[idx+1];
                    in[idx+1]=temp;
                    swapedeven = true;
                }
            }
        }
        __syncthreads();
        if (!(swapedeven||swapedodd))
        break;
        odd=!odd;
    }
    __syncthreads();
    
    if (idx < size)
        out[idx]=in[idx];
}

int main(void){
    int i;
    int *a,*a_sorted;
    int *d_a,*d_sorted;
    int blocks=1;
    int threads = 1024;
    int n=blocks*threads;
    int size =sizeof(int) * n;
    hipMalloc((void**) &d_a,size);
    hipMalloc((void**) &d_sorted,size);
    a = (int*) malloc(size); random_ints(a,n);
    a_sorted = (int*) malloc(size);
   
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);  

    loop_inside<<<blocks,threads>>>(d_a,d_sorted,n);
    
    hipMemcpy(a_sorted,d_sorted,size,hipMemcpyDeviceToHost);
    for(i=0;i<n;i++){
        printf("%d\t",a_sorted[i]);
    }
    return 0;
}