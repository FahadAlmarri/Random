
#include <hip/hip_runtime.h>
#include <stdio.h>
void random_ints(int* a, int N)
{
   int i;
   for (i = 0; i < N; ++i)
    a[i] = rand();
}
__global__ void rankSort(int * array, int * result, int k) {
	int a = k / gridDim.x; 
	int b = k / blockDim.x; 
	__shared__ int tamBlocks; 
	__shared__ int tamThreads; 
	__shared__ int miNumero; 
	__shared__ int rank; 
	int localRank;
	int comparador;
	int range2 = threadIdx.x * b; 
	if(threadIdx.x == 0) {
		tamBlocks = k - (a * gridDim.x);
		tamThreads = k - (b * blockDim.x);
	}
	int range1 = blockIdx.x * a; 
	for(int i = range1; i < range1 + a; i++) {	
		if(threadIdx.x == 0) {
			miNumero = array[i]; 
			rank = 0; 
		}
		__syncthreads();
		localRank = 0; 
		for(int j = range2; j < range2 + b; j++) {
			comparador = array[j];
			if(comparador < miNumero || (comparador == miNumero && (j < i)))
				localRank += 1; 
		}
		if(threadIdx.x < tamThreads) { 
			comparador = array[(blockDim.x * b) + threadIdx.x];
			if(comparador < miNumero || (comparador == miNumero && (((blockDim.x * b) + threadIdx.x) < i)))
				localRank += 1;
		}
		atomicAdd(&rank, localRank); 
		__syncthreads();
		if(threadIdx.x == 0) {
			result[rank] = miNumero; 
		}
		__syncthreads();
	}
	if(blockIdx.x < tamBlocks) {
		if(threadIdx.x == 0) {
			miNumero = array[gridDim.x * a  + blockIdx.x];
			rank = 0;
		}
		__syncthreads();
		localRank = 0;
		for(int j = range2; j < range2 + b; j++) {
			comparador = array[j];
			if(comparador < miNumero || (comparador == miNumero && (j < (gridDim.x * a + blockIdx.x))))
				localRank += 1; 
		}
		if(threadIdx.x < tamThreads) {
			comparador = array[(blockDim.x * b) + threadIdx.x];
			if(comparador < miNumero || (comparador == miNumero && (((blockDim.x * b) + threadIdx.x) < gridDim.x * a  + blockIdx.x)))
				localRank += 1; 
		}
		atomicAdd(&rank, localRank); 
		__syncthreads();
		if(threadIdx.x == 0) {
			result[rank] = miNumero; 
		}
	}
}int main(void){
    int i;
    int *a,*a_sorted;
    int *d_a,*d_sorted;
    int blocks=128;
    int threads = 1024;
    int n=blocks*threads;
    int size =sizeof(int) * n;
    hipMalloc((void**) &d_a,size);
    hipMalloc((void**) &d_sorted,size);
    a = (int*) malloc(size); random_ints(a,n);
    a_sorted = (int*) malloc(size);
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);  
    rankSort<<<blocks,threads>>>(d_a,d_sorted,n);  
    hipMemcpy(a_sorted,d_sorted,size,hipMemcpyDeviceToHost);
    for(i=0;i<n;i++){
        printf("%d\t",a_sorted[i]);
    }
    return 0;
}